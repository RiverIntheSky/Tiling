#include <hip/hip_runtime.h>
#include <>

#include <iostream>

#include "WFObjectToString.h"
#include "UniformGridSortBuilderTest.h"
#include "GraphTest.h"
#include "CollisionTest.h"
#include "ShapeVariationTest.h"
#include "RNGTest.h"
#include "WiggleTest.h"

#include <thrust/detail/config.h>

int main()
{

#if 0//THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n";
		return 1;
	}
#endif

	RNGTest rngTest;
	int rng_test_result = rngTest.testAll();
	//graphTest.testAll(1000);

	if (rng_test_result != 0)
	{
		std::cerr << "Random number generator test failed!\n";
		return rng_test_result;
	}
	else
	{
		std::cerr << "Random number generator test passed.\n";
	}

	UniformGridSortBuildTest uniformGridTest;
	int ugrid_test_result = uniformGridTest.testAll("../scenes/church/church.obj", 32, 16, 24);
	
	if (ugrid_test_result != 0)
	{
		std::cerr << "Uniform grid construction test failed!\n";
		return ugrid_test_result;
	}
	else
	{
		std::cerr << "Uniform grid construction test passed.\n";

	}
	
	GraphTest graphTest;
	int graph_test_result = graphTest.testAll(1000);
	//graphTest.testAll(1000);
	
	if (graph_test_result != 0)
	{
		std::cerr << "Graph construction test failed!\n";
		return graph_test_result;
	}
	else
	{
		std::cerr << "Graph construction test passed.\n";
	}
	
	CollisionTest collTest;
	int coll_test_result = collTest.testAll("../scenes/castle/castle.obj");
	if (coll_test_result != 0)
	{
		std::cerr << "Collision detection test failed!\n";
		return coll_test_result;
	}
	else
	{
		std::cerr << "Collision detection test passed. \n";
	}
	
	std::cerr << "---------------------------------------------------------------------\n";
	const char* obj2strTestFile = "../scenes/church/church.obj";
	std::cerr << obj2strTestFile << " converted to \n"
		<< WFObjectToString(obj2strTestFile) << "\n";


	const char* variationFile1 = "../scenes/test_church/c19.obj";
	const char* variationFile2 = "../scenes/test_church/c28.obj";
	//const char* variationFile1 = "../scenes/test_skyscraper/v01.obj";
	//const char* variationFile2 = "../scenes/test_skyscraper/v02.obj";
	//const char* variationFile1 = "../scenes/test_sand_castle/v01.obj";
	//const char* variationFile2 = "../scenes/test_sand_castle/v02.obj";
	//const char* variationFile1 = "../scenes/test_playground/v001.obj";
	//const char* variationFile2 = "../scenes/test_playground/v002.obj";
	//const char* variationFile1 = "../scenes/moon_base/variant_01.obj";
	//const char* variationFile2 = "../scenes/moon_base/variant_02.obj";

	ShapeVariationTest variationTest;
	int var_test_result = variationTest.testAll(variationFile1, variationFile2);
	if (var_test_result != 0)
	{
		std::cerr << "Shape variation test failed!\n";
		return var_test_result;
	}
	else
	{
		std::cerr << "Shape variation test passed. \n";
	}


	//const char* wiggleFile1 = "../scenes/wiggle_test/v001.obj";
	//const char* wiggleFile2 = "../scenes/wiggle_test/v002.obj";
	//const char* wiggleFile3 = "../scenes/wiggle_test/v001_v002_2_v002_2_v001_v002_3_v002_3_2_v001_v002_3_v002_3_v002_5_1.obj";
	//const char* wiggleFile3 = "../scenes/wiggle_test/v001_v001_v002_2_3_v001_v002_3_v002_3_3_v001_v002_2_v002_2_v001_v002_3_v002_3_3_3.obj";	
	//const char* wiggleFile3 = "../scenes/wiggle_test/v002.obj";

	const char* wiggleFile1 = "../scenes/wiggle_test/c19.obj";
	const char* wiggleFile2 = "../scenes/wiggle_test/c28.obj";
	//const char* wiggleFile3 = "../scenes/wiggle_test/v_1_4_11.obj";
	//const char* wiggleFile3 = "../scenes/wiggle_test/c19_c19_c28_12_1.obj";
	const char* wiggleFile3 = "../scenes/wiggle_test/v_1_4_12.obj";

	//const char* wiggleFile1 = "../scenes/skyscraper/v01.obj";
	//const char* wiggleFile2 = "../scenes/skyscraper/v02.obj";
	//const char* wiggleFile3 = "../scenes/skyscraper/v01_v02_5.obj";

	//const char* wiggleFile1 = "../scenes/sand_castle/v01.obj";
	//const char* wiggleFile2 = "../scenes/sand_castle/v02.obj";
	//const char* wiggleFile3 = "../scenes/wiggle_test/v01_v02_3.obj";


	const char* wiggleOutFile = "../scenes/wiggle_test/fixed.obj";

	WiggleTest wiggleTest;
	int wiggle_test_result = wiggleTest.testAll(wiggleFile1, wiggleFile2, wiggleFile3, wiggleOutFile);
	if (wiggle_test_result != 0)
	{
		std::cerr << "Wiggle test failed!\n";
		if (wiggle_test_result == 1)
		{
			std::cerr << "Invalid repair target - does not conform grammar.\n";
		}
		else if (wiggle_test_result == 2)
		{
			std::cerr << "Object repair attempt failed.\n";
		}
		return wiggle_test_result;
	}
	else
	{
		std::cerr << "Object repair attempt succeeded.\n";
		std::cerr << "Wrote " << wiggleOutFile << "\n";
	}


#if 0//THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
     //hipDeviceReset must be called before exiting in order for profiling and
     //tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
		std::cerr << "hipDeviceReset failed!\n";
        return 1;
    }
#endif

    return 0;
}

